# include <iostream>
# include <cstdlib>
# include <string>
# include <chrono>
# include <cmath>
# include <vector>
# include <fstream>
#include <hip/hip_runtime.h>

/** Une structure complexe est définie pour la bonne raison que la classe
 * complex proposée par g++ est très lente ! Le calcul est bien plus rapide
 * avec la petite structure donnée ci--dessous
 **/
struct Complex
{
    Complex() : real(0.), imag(0.)
    {}
    Complex(double r, double i) : real(r), imag(i)
    {}
    Complex operator + ( const Complex& z )
    {
        return Complex(real + z.real, imag + z.imag );
    }
    Complex operator * ( const Complex& z )
    {
        return Complex(real*z.real-imag*z.imag, real*z.imag+imag*z.real);
    }
    double sqNorm() { return real*real + imag*imag; }
    double real,imag;
};

std::ostream& operator << ( std::ostream& out, const Complex& c )
{
  out << "(" << c.real << "," << c.imag << ")" << std::endl;
  return out;
}

__global__ void iterMandelbrot( int maxIter, float scaleX, float scaleY, int W, int H, int* image )
{
    unsigned int indx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int indy = threadIdx.y + blockIdx.y * blockDim.y;

    if ( (indx < W) && (indy < H) )
    {
        unsigned int glob_idx = indx + indy*W;
        float cr = -2.f+indx*scaleX;
        float ci = -1.125f+ indy*scaleY;
        float zr = 0.f;
        float zi = 0.f;

        int niter = 0;
        while ((zr*zr+zi*zi < 4.) && (niter < maxIter))
        {
            float zt = zr*zr-zi*zi + cr;
            zi = 2*zr*zi + ci;
            zr = zt;
            ++niter;
        }
        image[glob_idx] = niter;
    }
}

std::vector<int>
computeMandelbrotSet( int W, int H, int maxIter )
{
    std::chrono::time_point<std::chrono::system_clock> start, end;
    std::chrono::time_point<std::chrono::system_clock> start2, end2;
    std::vector<int> pixels(W*H);
    float scaleX = 3.f/(W-1.f);
    float scaleY = 2.25f/(H-1.f);
    int* d_pixel;
    start = std::chrono::system_clock::now();
    start2 = std::chrono::system_clock::now();
    hipMalloc(&d_pixel, W*H*sizeof(int));
    dim3 block{16,16,1};
    dim3 grid {unsigned((W+15)/16),unsigned((H+15)/16),1};
    iterMandelbrot<<<grid,block>>>(maxIter, scaleX, scaleY, W, H, d_pixel);

    hipMemcpy(pixels.data(), d_pixel, W*H*sizeof(int), hipMemcpyDeviceToHost);
    end2 = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds2 = end2-start2;
    std::cout << "Temps calcul noyau cuda ensemble mandelbrot : " << elapsed_seconds2.count() 
              << std::endl;

    hipFree(d_pixel);
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    std::cout << "Temps calcul ensemble mandelbrot : " << elapsed_seconds.count() 
              << std::endl;
    return pixels;
}

/** Construit et sauvegarde l'image finale **/
void savePicture( const std::string& filename, int W, int H, const std::vector<int>& nbIters, int maxIter )
{
    double scaleCol = 1./maxIter;//16777216
    std::ofstream ofs( filename.c_str(), std::ios::out | std::ios::binary );
    ofs << "P6\n"
        << W << " " << H << "\n255\n";
    for ( int i = 0; i < W * H; ++i ) {
        double iter = scaleCol*nbIters[i];
        unsigned char r = (unsigned char)(256 - (unsigned (iter*256.) & 0xFF));
        unsigned char b = (unsigned char)(256 - (unsigned (iter*65536) & 0xFF));
        unsigned char g = (unsigned char)(256 - (unsigned( iter*16777216) & 0xFF));
        ofs << r << g << b;
    }
    ofs.close();
}

int main(int argc, char *argv[] ) 
 { 
    const int W = 800;
    const int H = 600;
    int* d_tmp;
    hipMalloc(&d_tmp, 10*sizeof(int));
 
    // Normalement, pour un bon rendu, il faudrait le nombre d'itérations
    // ci--dessous :
    //const int maxIter = 16777216;
    const int maxIter = 8*65536;
    auto iters = computeMandelbrotSet( W, H, maxIter );
    savePicture("mandelbrot.tga", W, H, iters, maxIter);
    hipFree(d_tmp);
    return EXIT_SUCCESS;
 }
    
